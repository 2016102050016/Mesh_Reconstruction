#include "hip/hip_runtime.h"

#include "quadprog.h"
#include <hip/hip_runtime.h>

using namespace std;

#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                           const char *file,
                           int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

//more formatting for rapid algorithm
//takes advantage of sparseness
vector<int> getIr(SparseMatrixPtr R){
    vector<int> ir (R->nonZeros());
    int count=0;
    for (int k=0; k<R->outerSize(); ++k){
        for (Eigen::SparseMatrix<float>::InnerIterator it(*R,k); it; ++it){
            ir[count] = it.row();
            count++;
        }
    }
    return ir;

}

vector<int> getJc(SparseMatrixPtr R){
    vector<int> jc ((R->cols())+1);
    jc[0]=0;
    int sum=0;
    for (int k=0; k<R->outerSize(); ++k){
        for (Eigen::SparseMatrix<float>::InnerIterator it(*R,k); it; ++it){
            sum++;
        }
        jc[k+1]=sum;
    }
    return jc;
}

vector<float> getPr(SparseMatrixPtr R){
    vector<float> pr (R->nonZeros(), 0);
    int count=0;
    for (int k=0; k<R->outerSize(); ++k){
        for (Eigen::SparseMatrix<float>::InnerIterator it(*R,k); it; ++it){
            pr[count]=it.value();
            count++;
        }
    }
    return pr;
}

__global__ void doStepDevice(
    float *in,
    float *out,
    int* ir,
    int* jc,
    float* pr,
    float* invdg,
    float* lb,
    float* ub,
    int nRows
)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    if(row < nRows) {

        float res = 0;
        int start = jc[row];
        int end = jc[row + 1];
        int i;

        for (i = start; i < end; i++)
            res += pr[i] * in[ir[i]];

        res = (in[row] - res * invdg[row]) * 0.5;
        if (res < lb[row]) res = lb[row];
        if (res > ub[row]) res = ub[row];
        out[row] = res;
    }
}

//row increment
void doStep(int row, const vector<float>& in, vector<float>& out,
    const vector<int>& ir, const vector<int>& jc, const vector<float>& pr, const vector<float>& invdg,
    const vector<float>& lb, const vector<float>& ub)
{
    float res = 0;
    int start = jc[row];
    int end   = jc[row+1];
    int i;

    for(i = start; i < end; i++)
        res +=  pr[i]*in[ir[i]];

    res = (in[row]-res*invdg[row])*0.5;
    if(res < lb[row]) res = lb[row];
    if(res > ub[row]) res = ub[row];
    out[row] = res;
}

vector<float> runQPGPU(qp_argsPtr args,
                       const vector<int> &featureIndexes,
                       const bool USING_FEATURES)
{
    //get ir, jc, and pr
    vector<int> ir = getIr(args->R);
    vector<int> jc = getJc(args->R);
    vector<float> pr = getPr(args->R);

    int size = args->R->rows();

    vector<float>* out = new vector<float>(size, 0);

    int *irCu, *jcCu;
    float *prCu, *invdgCu, *lbCu, *ubCu, *inCu, *outCu;

    cout << "Creating memory spaces on GPU" << endl;
    GPU_CHECKERROR(hipMalloc((void**)&inCu,    size * sizeof(float)));
    GPU_CHECKERROR(hipMalloc((void**)&outCu,   size * sizeof(float)));
    GPU_CHECKERROR(hipMalloc((void**)&irCu,    ir.size() * sizeof(int)));
    GPU_CHECKERROR(hipMalloc((void**)&jcCu,    jc.size() * sizeof(int)));
    GPU_CHECKERROR(hipMalloc((void**)&prCu,    pr.size() * sizeof(float)));
    GPU_CHECKERROR(hipMalloc((void**)&invdgCu, args->invdg.size() * sizeof(float)));
    GPU_CHECKERROR(hipMalloc((void**)&lbCu,    args->lb.size() * sizeof(float)));
    GPU_CHECKERROR(hipMalloc((void**)&ubCu,    args->ub.size() * sizeof(float)));

    cout << "Copying memory to GPU" << endl;
    GPU_CHECKERROR(hipMemcpy(inCu, &(args->x)[0],        size * sizeof(float), hipMemcpyHostToDevice));
    GPU_CHECKERROR(hipMemcpy(irCu, &ir[0],               ir.size() * sizeof(int), hipMemcpyHostToDevice));
    GPU_CHECKERROR(hipMemcpy(jcCu, &jc[0],               jc.size() * sizeof(int), hipMemcpyHostToDevice));
    GPU_CHECKERROR(hipMemcpy(prCu, &pr[0],               pr.size() * sizeof(float), hipMemcpyHostToDevice));
    GPU_CHECKERROR(hipMemcpy(invdgCu, &(args->invdg)[0], args->invdg.size() * sizeof(float), hipMemcpyHostToDevice));
    GPU_CHECKERROR(hipMemcpy(lbCu, &(args->lb)[0],       args->lb.size() * sizeof(float), hipMemcpyHostToDevice));
    GPU_CHECKERROR(hipMemcpy(ubCu, &(args->ub)[0],       args->ub.size() * sizeof(float), hipMemcpyHostToDevice));

    int BLOCK_SIZE = 256;
    int NBLOCKS = (int)ceil(double(size)/BLOCK_SIZE);

    //perform algorithm
    cout << "Perform GPU algorithm" << endl;
    for(int i = 0; i < args->iter; i++){
        if(i > 0) {
            float* swap = inCu;
            inCu = outCu;
            outCu = swap;
        }
        doStepDevice<<<NBLOCKS,BLOCK_SIZE>>>(inCu, outCu, irCu, jcCu, prCu, invdgCu, lbCu, ubCu, size);
//        hipMemcpy(inCu, outCu, size * sizeof(float), hipMemcpyDeviceToDevice);
    }

    hipDeviceSynchronize();

    hipMemcpy(&((*out)[0]), outCu, size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    cout << "Free all GPU Memory" << endl;
    hipFree(irCu);
    hipFree(jcCu);
    hipFree(prCu);
    hipFree(invdgCu);
    hipFree(lbCu);
    hipFree(ubCu);
    hipFree(inCu);
    hipFree(outCu);

    if(USING_FEATURES){
        //reset values at feature points
        int count=0;
        for(int i=0; i<args->x.size(); i++){
            if(i==featureIndexes[count]){
                (*out)[i]=0.2; //<--------------------------------------could be related to band size
                count++;
            }
        }
    }

    cout<<"quadratic program finished"<<endl;
    return *out;
}

//quadratic programming optimization algorithm
vector<float> runQP(qp_argsPtr args, const vector<int> &featureIndexes, const bool USING_FEATURES){
    //get ir and jc
    vector<int> ir = getIr(args->R);
    vector<int> jc = getJc(args->R);
    vector<float> pr = getPr(args->R);

    int size = args->R->rows();

    vector<float>* buf1 = new vector<float>(args->x);
    vector<float>* buf2 = new vector<float>(size, 0);
    vector<float>* in;
    vector<float>* out;

    //perform algorithm
    for(int i = 0; i < args->iter; i++){
        if(i % 2){
            in = buf2;
            out = buf1;
        }
        else{
            in = buf1;
            out = buf2;
        }

        for(int r = 0; r < size; r++){
            doStep(r, *in, *out, ir, jc, pr, args->invdg, args->lb, args->ub);
        }
    }
    if(USING_FEATURES){
        //reset values at feature points
        int count=0;
        for(int i=0; i<args->x.size(); i++){
            if(i==featureIndexes[count]){
                (*out)[i]=0.2; //<--------------------------------------could be related to band size
                count++;
            }
        }
    }

    cout<<"quadratic program finished"<<endl;
    return *out;
}



//Function for computing weighted voxel grid for marching cubes
//takes as input a binary volume
gridPtr optimize(gridPtr volume, gridPtr featureMap, const bool USING_FEATURES, const bool USING_CUDA){
    int BAND_SIZE=4.0;
    //prime quadratic programming arguments
    //prepare margin
    gridPtr margin;
    if(USING_CUDA)
        margin = dfield_gpu(volume);
    else
        margin = getsqrt(getsqdist(fastPerim(volume)));
    cout<<"margin calculated"<<endl;
    //prepare bands
    bandsPtr bnds = createBands(margin, BAND_SIZE);
    cout<<"bands created"<<endl;
    //get band point indexes
    vector<int> indexes = findIndexes(bnds->band);
    //create index map
    gridPtr indexMap = getIndexMap(bnds->band, indexes);
    vector<int> featureIndexes = getFeatureIndexes(featureMap, indexMap);
    cout<<"indexes stored"<<endl;

    //prepare qp_args
    qp_argsPtr args = primeQP(volume, margin, bnds);

    //run quadratic programming
    vector<float> x;
    if(USING_CUDA)
        x = runQPGPU(args, featureIndexes, USING_FEATURES);
    else
        x = runQP(args, featureIndexes, USING_FEATURES);

    //prepare new voxel grid with embedding function
    gridPtr F = copyGrid(volume);
    for(int i=0; i<F->dims[0]; i++){
        for(int j=0; j<F->dims[1]; j++){
            for(int k=0; k<F->dims[2]; k++){
                (*F)[i][j][k]=(2.0*(*F)[i][j][k]-1.0)*(BAND_SIZE+1.0);
            }
        }
    }
    for(int i=0; i<indexes.size(); i++){
        (*F)(indexes[i]) = x[i];
    }

    return F;
}
