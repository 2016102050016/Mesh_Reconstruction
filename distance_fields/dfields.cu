#include "hip/hip_runtime.h"

#include "dfields.h"

using namespace std;

#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                           const char *file,
                           int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define getFlatIndex(x,y,z,xdim,ydim,zdim) z*ydim*xdim+y*xdim+x

#define get3DIndex(i,j,k,index,xdim,ydim,zdim) k=index/(xdim*ydim),index-=k*xdim*ydim,j=index/xdim,index-=j*xdim,i=index/1

float* flatten(gridPtr g) {
    float* g_flat = new float[g->dims[0] * g->dims[1] * g->dims[2]];
    for(int i=0; i<g->dims[0]; i++){
        for(int j=0; j<g->dims[1]; j++){
            for(int k=0; k<g->dims[2]; k++){
                g_flat[getFlatIndex(i,j,k,g->dims[0],g->dims[1],g->dims[2])] = (*g)[i][j][k];
            }
        }
    }
    return g_flat;
}

gridPtr unflatten(float* g_flat, gridPtr volume_grid, int xdim, int ydim, int zdim) {
    gridPtr g(new grid(volume_grid->dims, volume_grid->scale, volume_grid->shift, volume_grid->pad));
    for(int i=0; i<xdim; i++) {
        for (int j = 0; j < ydim; j++) {
            for (int k = 0; k < zdim; k++) {
                (*g)[i][j][k] = g_flat[getFlatIndex(i,j,k,xdim,ydim,zdim)];
            }
        }
    }
    return g;
}

int numDiff(float *g, gridPtr real) {
    int numDiff = 0;
    for(int i=0; i<real->dims[0]; i++) {
        for (int j = 0; j < real->dims[1]; j++) {
            for (int k = 0; k < real->dims[2]; k++) {
                if((*real)[i][j][k] != g[getFlatIndex(i,j,k,real->dims[0],real->dims[1],real->dims[2])]) {
                    numDiff++;
                    cout << (*real)[i][j][k] << "," << g[getFlatIndex(i,j,k,real->dims[0],real->dims[1],real->dims[2])] << " | ";
                }
            }
        }
    }
    cout << endl;
    return numDiff;
}

__global__ void fastPerimGPU(float* src_grid, float* dest_grid, int xdim, int ydim, int zdim) {
    int center = blockIdx.x*blockDim.x+threadIdx.x;

    if(center < xdim * ydim * zdim) {
        dest_grid[center] = 1.0;
        int i,j,k,index=center;
        get3DIndex(i,j,k,index,xdim,ydim,zdim);

        int neg_i=i>0;
        int neg_j=j>0;
        int neg_k=k>0;
        int pos_i=i<xdim-1;
        int pos_j=j<ydim-1;
        int pos_k=k<zdim-1;

        float value = src_grid[center];
        for(int i_=i-neg_i; i_<=i+pos_i; i_++){
            for(int j_=j-neg_j; j_<=j+pos_j; j_++){
                for(int k_=k-neg_k; k_<=k+pos_k; k_++){
                    if(value!=src_grid[getFlatIndex(i_,j_,k_, xdim,ydim,zdim)]) {
                        dest_grid[center]=0.0;
                        return;
                    }
                }
            }
        }
    }
}

__global__ void sqdistpt1(float* dist_grid, float* dest_grid, int xdim, int ydim, int zdim) {
    int center = blockIdx.x*blockDim.x+threadIdx.x;

    if(center < xdim * ydim * zdim) {

        int i,j,k,var=center;
        get3DIndex(i,j,k,var,xdim,ydim,zdim);

        //get squared dist to closest 0 in row (i's)
        float min = 100000;
        for(int index=0; index<xdim; index++){
            float dist = 100000;
            if(dist_grid[getFlatIndex(index, j, k, xdim, ydim, zdim)]==0.0){
                dist = (index-i)*(index-i);
            }
            if(dist<min){
                min=dist;
            }
        }
        //set value of voxel to min q dist
        dest_grid[center]=(float)min;
    }
}

__global__ void sqdistpt2(float* dist_grid, float* dest_grid, int xdim, int ydim, int zdim) {
    int center = blockIdx.x*blockDim.x+threadIdx.x;

    if(center < xdim * ydim * zdim) {

        int i,j,k,var=center;
        get3DIndex(i,j,k,var,xdim,ydim,zdim);

        //get squared dist to closest 0 in row (i's)
        int min = 100000;
        for(int index=0; index<ydim; index++){
            int dist = dist_grid[getFlatIndex(i, index, k, xdim, ydim, zdim)] + (index - j) * (index - j);
            if(dist<min) min=dist;
        }
        //set value of voxel to min q dist
        dest_grid[center]=(float)min;
    }
}

__global__ void sqdistpt3(float* dist_grid, float* dest_grid, int xdim, int ydim, int zdim) {
    int center = blockIdx.x*blockDim.x+threadIdx.x;

    if(center < xdim * ydim * zdim) {

        int i,j,k,var=center;
        get3DIndex(i,j,k,var,xdim,ydim,zdim);

        //get squared dist to closest 0 in row (i's)
        int min = 100000;
        for(int index=0; index<zdim; index++){
            int dist = dist_grid[getFlatIndex(i, j, index, xdim, ydim, zdim)] + (index - k) * (index - k);
            if(dist<min) min=dist;
        }
        //set value of voxel to min q dist
        dest_grid[center]=(float)min;
    }
}

__global__ void sqrtgpu(float* dist_grid, float* dest_grid, int xdim, int ydim, int zdim) {
    int center = blockIdx.x*blockDim.x+threadIdx.x;

    if(center < xdim * ydim * zdim) {
        dest_grid[center] = sqrt(dist_grid[center]);
    }
}

gridPtr dfield_gpu(gridPtr volume_grid) {
    float* flat_g = flatten(volume_grid);
    int xdim = volume_grid->dims[0], ydim = volume_grid->dims[1], zdim = volume_grid->dims[2];
    int g_size = xdim * ydim * zdim;

    float* dest_grid, *src_grid;
    cout << "Initializing dfields arrays" << endl;
    GPU_CHECKERROR(hipMalloc((void**) &dest_grid, g_size * sizeof(float)));
    GPU_CHECKERROR(hipMalloc((void**) &src_grid, g_size * sizeof(float)));

    GPU_CHECKERROR(hipMemcpy(src_grid, flat_g, g_size * sizeof(float), hipMemcpyHostToDevice));

    int BLOCK_SIZE = 256;
    int NUM_BLOCKS = (int)ceil(double(g_size)/BLOCK_SIZE);

    //Alternate buffers so we don't have to move data around on the device
    //Buffers effectively alternate between input and output
    cout << "dfield Fast perim gpu" << endl;
    fastPerimGPU<<<NUM_BLOCKS, BLOCK_SIZE>>>(src_grid, dest_grid, xdim, ydim, zdim);

    cout << "dfield sqdist" << endl;
    sqdistpt1<<<NUM_BLOCKS, BLOCK_SIZE>>>(dest_grid, src_grid, xdim, ydim, zdim);
    sqdistpt2<<<NUM_BLOCKS, BLOCK_SIZE>>>(src_grid, dest_grid, xdim, ydim, zdim);
    sqdistpt3<<<NUM_BLOCKS, BLOCK_SIZE>>>(dest_grid, src_grid, xdim, ydim, zdim);

    cout << "dfield sqrt" << endl;
    sqrtgpu<<<NUM_BLOCKS, BLOCK_SIZE>>>(src_grid, dest_grid, xdim, ydim, zdim);

    //Copy data back to device
    GPU_CHECKERROR(hipMemcpy(flat_g, dest_grid, g_size * sizeof(float), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    hipFree(dest_grid);
    hipFree(src_grid);

    hipDeviceSynchronize();

    return unflatten(flat_g, volume_grid, xdim, ydim, zdim);
}

//extract perimeter of binary volume
gridPtr fastPerim(gridPtr volume_grid){
    gridPtr g(new grid(volume_grid->dims, volume_grid->scale, volume_grid->shift, volume_grid->pad));
    for(int i=0; i<g->dims[0]; i++){
        for(int j=0; j<g->dims[1]; j++){
            for(int k=0; k<g->dims[2]; k++){
                (*g)[i][j][k]=1.0;
                int neg_i=1;int neg_j=1;int neg_k=1;
                int pos_i=1; int pos_j=1; int pos_k=1;
                if(i==0)neg_i=0;
                else if(i==g->dims[0]-1)pos_i=0;
                if(j==0)neg_j=0;
                else if(j==g->dims[1]-1)pos_j=0;
                if(k==0)neg_k=0;
                else if(k==g->dims[2]-1)pos_k=0;
                //search neighboring voxels for different value
                for(int i_=i-neg_i; i_<=i+pos_i; i_++){
                    for(int j_=j-neg_j; j_<=j+pos_j; j_++){
                        for(int k_=k-neg_k; k_<=k+pos_k; k_++){
                            if((*volume_grid)[i][j][k]!=(*volume_grid)[i_][j_][k_]){
                                (*g)[i][j][k]=0.0;
                            }
                        }
                    }
                }
                //end of neighbor search
            }
        }
    }

    return g;
}

//implementation of Saito and Toriwaki distance field
//squared euclidean distance to closest 0-value voxel
gridPtr getsqdist(gridPtr volume_grid){
    //make copies
    gridPtr dist_grid = copyGrid(volume_grid);
    gridPtr dist_grid_copy = copyGrid(dist_grid);
    //first transformation
    for(int i=0; i<dist_grid->dims[0]; i++){
        for(int j=0; j<dist_grid->dims[1]; j++){
            for(int k=0; k<dist_grid->dims[2]; k++){
                //get squared dist to closest 0 in row (i's)
                float min = 100000;
                for(int index=0; index<dist_grid->dims[0]; index++){
                    float dist = 100000;
                    if((*dist_grid)[index][j][k]==0.0){
                        dist = (index-i)*(index-i);
                    }
                    if(dist<min){
                        min=dist;
                    }
                }
                //set value of voxel to min q dist
                (*dist_grid_copy)[i][j][k]=min;
            }
        }
    }//end of first transformation

    dist_grid=copyGrid(dist_grid_copy);
    //second transformation
    for(int i=0; i<dist_grid->dims[0]; i++){
        for(int j=0; j<dist_grid->dims[1]; j++){
            for(int k=0; k<dist_grid->dims[2]; k++){
                int min = 100000;
                for(int index=0; index<dist_grid->dims[1]; index++){
                    int dist = (*dist_grid)[i][index][k] + (index-j)*(index-j);
                    if(dist<min) min=dist;
                }
                //set value of voxel to min q dist
                (*dist_grid_copy)[i][j][k]=(float)min;
            }
        }
    }//end of second transformation

    dist_grid=copyGrid(dist_grid_copy);
    //third transformation
    for(int i=0; i<dist_grid->dims[0]; i++){
        for(int j=0; j<dist_grid->dims[1]; j++){
            for(int k=0; k<dist_grid->dims[2]; k++){
                int min = 100000;
                for(int index=0; index<dist_grid->dims[2]; index++){
                    int dist = (*dist_grid)[i][j][index] + (index-k)*(index-k);
                    if(dist<min) min=dist;
                }
                //set value of voxel to min q dist
                (*dist_grid_copy)[i][j][k]=(float)min;
            }
        }
    }//end of third transformation

    return dist_grid_copy;
}

//returns linear indexes of closest 0-value voxel
gridPtr getsqdist_index(gridPtr volume_grid){
    //make copies
    gridPtr index_grid;
    gridPtr index_grid_copy = copyGrid(volume_grid);
    gridPtr dist_grid = copyGrid(volume_grid);
    gridPtr dist_grid_copy = copyGrid(dist_grid);
    //first transformation
    for(int i=0; i<dist_grid->dims[0]; i++){
        for(int j=0; j<dist_grid->dims[1]; j++){
            for(int k=0; k<dist_grid->dims[2]; k++){
                //get squared dist to closest 0 in row (i's)
                float min = 100000;
                float x_ind = -1.0;
                for(int index=0; index<dist_grid->dims[0]; index++){
                    float dist = 100000;
                    if((*dist_grid)[index][j][k]==0.0){
                        dist = (index-i)*(index-i);
                    }
                    if(dist<min){
                        min=dist;
                        x_ind = index;
                    }
                }
                //set value of voxel to min q dist
                (*dist_grid_copy)[i][j][k]=min;
                //set index value to x index
                (*index_grid_copy)[i][j][k]=x_ind;
            }
        }
    }//end of first transformation

    index_grid=copyGrid(index_grid_copy);
    dist_grid=copyGrid(dist_grid_copy);
    //second transformation
    for(int i=0; i<dist_grid->dims[0]; i++){
        for(int j=0; j<dist_grid->dims[1]; j++){
            for(int k=0; k<dist_grid->dims[2]; k++){
                int min = 100000;
                float xy_ind = -1.0;
                for(int index=0; index<dist_grid->dims[1]; index++){
                    int dist = (*dist_grid)[i][index][k] + (index-j)*(index-j);
                    if(dist<min){
                        min=dist;
                        xy_ind = (float)(index*index_grid->dims[0])+(*index_grid)[i][index][k];
                    }
                }
                //set value of voxel to min q dist
                (*dist_grid_copy)[i][j][k]=(float)min;
                (*index_grid_copy)[i][j][k] = xy_ind;
            }
        }
    }//end of second transformation

    index_grid=copyGrid(index_grid_copy);
    dist_grid=copyGrid(dist_grid_copy);
    //third transformation
    for(int i=0; i<dist_grid->dims[0]; i++){
        for(int j=0; j<dist_grid->dims[1]; j++){
            for(int k=0; k<dist_grid->dims[2]; k++){
                int min = 100000;
                float xyz_ind=-1.0;
                for(int index=0; index<dist_grid->dims[2]; index++){
                    int dist = (*dist_grid)[i][j][index] + (index-k)*(index-k);
                    if(dist<min){
                        min=dist;
                        xyz_ind = (float)(index*index_grid->dims[0]*index_grid->dims[1])+(*index_grid)[i][j][index];
                    }
                }
                //set value of voxel to min q dist
                (*dist_grid_copy)[i][j][k]=(float)min;
                (*index_grid_copy)[i][j][k] = xyz_ind;
            }
        }
    }//end of third transformation

    return index_grid_copy;

}

//get square root of grid
gridPtr getsqrt(gridPtr g){
    gridPtr s(new grid(g->dims, g->scale, g->shift, g->pad));
    for(int i=0; i<s->dims[0]; i++){
        for(int j=0; j<s->dims[1]; j++){
            for(int k=0; k<s->dims[2]; k++){
                (*s)[i][j][k]= (float) sqrt((double)(*g)[i][j][k]);
            }
        }
    }
    return s;
}


//*****************************************************************************************************
//functions for computing normals of surface

//get linear indices of neighboring voxels
vector<int> getNeighbors(gridPtr g, const Eigen::Vector3i &pnt){
    vector<int> indexes;
    for(int i=pnt[0]-1; i<=pnt[0]+1; i++){
        for(int j=pnt[1]-1; j<=pnt[1]+1; j++){
            for(int k=pnt[2]-1; k<=pnt[2]+1; k++){
                if((*g)[i][j][k]>0){
                    Eigen::Vector3i p;
                    p[0]=i; p[1]=j; p[2]=k;
                    indexes.push_back(g->sub2ind(p));
                }
            }
        }
    }
    return indexes;
}

//check if voxel is on surface
bool isSurface(gridPtr g, const Eigen::Vector3i &pnt){
    //search neighboring voxels for different value
    int i=pnt[0];
    int j=pnt[1];
    int k=pnt[2];
    if((*g)[i][j][k]!=1.0) return false;
    for(int i_=i-1; i_<=i+1; i_++){
        for(int j_=j-1; j_<=j+1; j_++){
            for(int k_=k-1; k_<=k+1; k_++){
                if((*g)[i_][j_][k_]==0.0){
                    return true;
                }
            }
        }
    }
    return false;
}

//get linear indices of neighboring voxels on the surface
vector<int> getSurfaceNeighbors(gridPtr g, const Eigen::Vector3i &pnt){
    vector<int> indexes;
    for(int i=pnt[0]-1; i<=pnt[0]+1; i++){
        for(int j=pnt[1]-1; j<=pnt[1]+1; j++){
            for(int k=pnt[2]-1; k<=pnt[2]+1; k++){
                if((*g)[i][j][k]>0){
                    Eigen::Vector3i p;
                    p[0]=i; p[1]=j; p[2]=k;
                    if(isSurface(g,p)){
                        indexes.push_back(g->sub2ind(p));
                    }
                }
            }
        }
    }
    return indexes;
}

//get centroid of a list of points
Eigen::Vector3f getCentroid(gridPtr g, const vector<int> &pnts){
    Eigen::Vector3f centroid;
    centroid[0]=0.0; centroid[1]=0.0; centroid[2]=0.0;
    for(int i=0; i<pnts.size(); i++){
        Eigen::Vector3i p = g->ind2sub(pnts[i]);
        centroid[0]+=(float)p[0]; centroid[1]+=(float)p[1]; centroid[2]+=(float)p[2];
    }
    centroid[0]=centroid[0]/(float)pnts.size();
    centroid[1]=centroid[1]/(float)pnts.size();
    centroid[2]=centroid[2]/(float)pnts.size();
    return centroid;
}

//get covariance matrix of surface point
Eigen::Matrix3f getCovariance(gridPtr g, const Eigen::Vector3i &pnt){
    vector<int> indexes = getSurfaceNeighbors(g, pnt);
    Eigen::Vector3f centroid = getCentroid(g, indexes);
    Eigen::Matrix3f covariance;
    covariance<<0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0;
    for(int i=0; i<indexes.size(); i++){
        Eigen::Vector3f vec;
        vec[0]=(float)(g->ind2sub(indexes[i]))[0] - centroid[0];
        vec[1]=(float)(g->ind2sub(indexes[i]))[1] - centroid[1];
        vec[2]=(float)(g->ind2sub(indexes[i]))[2] - centroid[2];
        for(int row=0; row<3; row++){
            for(int col=0; col<3; col++){
                covariance(row,col) = covariance(row,col)+vec[row]*vec[col];
            }
        }
    }
    for(int row=0; row<3; row++){
        for(int col=0; col<3; col++){
            covariance(row,col) = covariance(row,col)/(float)indexes.size();
        }
    }
    return covariance;
}

//get normal vector from covariance matrix
Eigen::Vector3f getNormalVector(const Eigen::Matrix3f &covariance){
    Eigen::SelfAdjointEigenSolver<Eigen::Matrix3f> es;
    es.compute(covariance);
    float min=10000;
    int index=0;
    for(int i=0; i<3; i++){
        int eigenvalue=es.eigenvalues()[i];
        if(eigenvalue<min){
            min=eigenvalue;
            index=i;
        }
    }
    Eigen::Vector3f vec = es.eigenvectors().col(index);
    float len = sqrt(vec[0]*vec[0]+vec[1]*vec[1]+vec[2]*vec[2]);
    vec[0]=vec[0]/len; vec[1]=vec[1]/len; vec[2]=vec[2]/len;
    return vec;
}

//orient normal vector with respect to local centroid
void orientNormal(gridPtr g, Eigen::Vector3f &normal, const Eigen::Vector3i &pnt){
    Eigen::Vector3f centroid = getCentroid(g, getNeighbors(g, pnt));
    centroid[0]-=(float)pnt[0]; centroid[1]-=(float)pnt[1]; centroid[2]-=(float)pnt[2];
    float prod = normal[0]*centroid[0]+normal[1]*centroid[1]+normal[2]*centroid[2];
    if(prod>0.0){
        normal[0]=-normal[0]; normal[1]=-normal[1]; normal[2]=-normal[2];
    }
}

//get indexes of surface points of volume grid
vector<int> getSurface(gridPtr volume){
    vector<int> indexes;
    gridPtr surface(new grid(volume->dims, volume->scale, volume->shift, volume->pad));
    for(int i=0; i<surface->dims[0]; i++){
        for(int j=0; j<surface->dims[1]; j++){
            for(int k=0; k<surface->dims[2]; k++){
                Eigen::Vector3i pnt;
                pnt[0]=i; pnt[1]=j; pnt[2]=k;
                if(isSurface(volume, pnt)){
                    indexes.push_back(volume->sub2ind(pnt));
                }
            }
        }
    }
    return indexes;
}

//get surface normals
vector<Eigen::Vector3f> getSurfaceNormals(gridPtr volume, const vector<int> &surface){
    vector<Eigen::Vector3f> normals;
    for(int i=0; i<surface.size(); i++){
        Eigen::Matrix3f covariance = getCovariance(volume, volume->ind2sub(surface[i]));
        Eigen::Vector3f norm = getNormalVector(covariance);
        normals.push_back(norm);
        orientNormal(volume, normals[i], volume->ind2sub(surface[i]));
    }
    return normals;
}

//returns index in vector of value, -1 if not contained
int contains(vector<int> vec, int val){
    for(int i=0; i<vec.size(); i++){
        if(vec[i]==val) return i;
    }
    return -1;
}


//create normals and visualize in pcl viewer
void visualizeNormals(gridPtr volume){
    vector<int> surface = getSurface(volume);
    gridPtr surfaceMap = getIndexMap(volume, surface);
    vector<Eigen::Vector3f> normals = getSurfaceNormals(volume, surface);

    //create point cloud from volume
    //convert imbedding function to point cloud for visualization
    //create point clouds from bands for visualization
    pcl::PointCloud<pcl::PointXYZRGB>::Ptr pcl_grid (new pcl::PointCloud<pcl::PointXYZRGB>());
    for(int i=0; i<volume->dims[0]; i++){
        for(int j=0; j<volume->dims[1]; j++){
            for(int k=0; k<volume->dims[2]; k++){
                if((*volume)[i][j][k]>0){
                    Eigen::Vector3i pnt;
                    pnt[0]=i; pnt[1]=j; pnt[2]=k;
                    int index = volume->sub2ind(pnt);
                    int ind = contains(surface, index);
                    if(ind==-1){
                        pcl::PointXYZRGB p;
                        p.x=(float)i; p.y=(float)j; p.z=(float)k;
                        p.r=0;p.g=0;p.b=255;
                        pcl_grid->push_back(p);
                    }
                    else{
                        pcl::PointXYZRGB p;
                        p.x=(float)i; p.y=(float)j; p.z=(float)k;
                        p.r=255;p.g=0;p.b=0;
                        pcl_grid->push_back(p);
                    }
                }
                else{
                    pcl::PointXYZRGB p;
                    p.x=(float)i; p.y=(float)j; p.z=(float)k;
                    p.r=0;p.g=0;p.b=0;
                    pcl_grid->push_back(p);
                }
            }
        }
    }

    //create point cloud of normals
    pcl::PointCloud<pcl::Normal>::Ptr normal_grid (new pcl::PointCloud<pcl::Normal>());
    for(int i=0; i<volume->dims[0]; i++){
        for(int j=0; j<volume->dims[1]; j++){
            for(int k=0; k<volume->dims[2]; k++){
                int ind = (*surfaceMap)[i][j][k];
                if(ind==-1){
                    pcl::Normal norm;
                    norm.normal_x = norm.normal_y = norm.normal_z = norm.data_n[3] = 0.0f;
                    norm.curvature = 0;
                    normal_grid->push_back(norm);
                }
                else{
                    pcl::Normal norm;
                    norm.normal_x = normals[ind][0]; norm.normal_y = normals[ind][1]; norm.normal_z = normals[ind][2];
                    norm.curvature = 0;
                    norm.data_n[3]=0.0f;
                    normal_grid->push_back(norm);
                }
            }
        }
    }

    //visualize
    boost::shared_ptr<pcl::visualization::PCLVisualizer> viewer (new pcl::visualization::PCLVisualizer ("3D Viewer"));
    viewer->setBackgroundColor (0, 0, 0);
    pcl::visualization::PointCloudColorHandlerRGBField<pcl::PointXYZRGB> rgb(pcl_grid);
    viewer->addPointCloud<pcl::PointXYZRGB> (pcl_grid, rgb, "sample cloud");
    viewer->setPointCloudRenderingProperties (pcl::visualization::PCL_VISUALIZER_POINT_SIZE, 3, "sample cloud");
    viewer->addPointCloudNormals<pcl::PointXYZRGB, pcl::Normal> (pcl_grid, normal_grid, 1, 0.25, "normals");
    viewer->addCoordinateSystem (1.0);
    viewer->initCameraParameters ();
    while (!viewer->wasStopped ())
    {
        viewer->spinOnce (100);
        boost::this_thread::sleep (boost::posix_time::microseconds (100000));
    }
}

//********************************************************************************************************************
//feature detection using normals

//input is binary volume pre-smoothing
//binary grid: 1=feature, 0=no feature
//features can be ignored during smoothing
//reasonable threshold ~0.9
gridPtr getFeatureMap(gridPtr volume, gridPtr surfaceMap, const vector<Eigen::Vector3f> &normals, float threshold){
    gridPtr featureMap (new grid(volume->dims, volume->scale, volume->shift, volume->pad));
    for(int i=0; i<featureMap->dims[0]; i++){
        for(int j=0; j<featureMap->dims[1]; j++){
            for(int k=0; k<featureMap->dims[2]; k++){
                (*featureMap)[i][j][k]=0.0;
            }
        }
    }

    int edgeTable[256]={
    0x0  , 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c,
    0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
    0x190, 0x99 , 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c,
    0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
    0x230, 0x339, 0x33 , 0x13a, 0x636, 0x73f, 0x435, 0x53c,
    0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
    0x3a0, 0x2a9, 0x1a3, 0xaa , 0x7a6, 0x6af, 0x5a5, 0x4ac,
    0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
    0x460, 0x569, 0x663, 0x76a, 0x66 , 0x16f, 0x265, 0x36c,
    0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
    0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0xff , 0x3f5, 0x2fc,
    0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
    0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x55 , 0x15c,
    0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
    0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0xcc ,
    0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
    0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc,
    0xcc , 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
    0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c,
    0x15c, 0x55 , 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
    0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc,
    0x2fc, 0x3f5, 0xff , 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
    0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c,
    0x36c, 0x265, 0x16f, 0x66 , 0x76a, 0x663, 0x569, 0x460,
    0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac,
    0x4ac, 0x5a5, 0x6af, 0x7a6, 0xaa , 0x1a3, 0x2a9, 0x3a0,
    0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c,
    0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x33 , 0x339, 0x230,
    0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c,
    0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x99 , 0x190,
    0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c,
    0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x0   };

    for(int i=0; i<volume->dims[0]-1; i++){
        for(int j=0; j<volume->dims[1]-1; j++){
            for(int k=0; k<volume->dims[2]-1; k++){
                //create gridcell
                int gridcell[8];
                for(int n=0; n<8; n++){
                    int i_ = ((n%4)==1 || (n%4)==2);
                    int j_ = (n%4)/2;
                    int k_ = n/4;
                    gridcell[n] = (int)(*volume)[i+i_][j+j_][k+k_];
                }
                int cubeindex=0;
                if (gridcell[0] < 1) cubeindex |= 1;
                if (gridcell[1] < 1) cubeindex |= 2;
                if (gridcell[2] < 1) cubeindex |= 4;
                if (gridcell[3] < 1) cubeindex |= 8;
                if (gridcell[4] < 1) cubeindex |= 16;
                if (gridcell[5] < 1) cubeindex |= 32;
                if (gridcell[6] < 1) cubeindex |= 64;
                if (gridcell[7] < 1) cubeindex |= 128;

                //check if cube is on the edge
                if(edgeTable[cubeindex]==0) continue;

                //check for feature in cube
                //get opening angles of normals
                float min=5.0;
                for(int n=0; n<7; n++){
                    for(int m=n+1; m<8; m++){
                        int i_1 = ((n%4)==1 || (n%4)==2);
                        int j_1 = (n%4)/2;
                        int k_1 = n/4;
                        int i_2 = ((m%4)==1 || (m%4)==2);
                        int j_2 = (m%4)/2;
                        int k_2 = m/4;
                        int index1 = (*surfaceMap)[i+i_1][j+j_1][k+k_1];
                        int index2 = (*surfaceMap)[i+i_2][j+j_2][k+k_2];
                        if(index1==-1 || index2==-1) continue;
                        Eigen::Vector3f norm1 = normals[index1];
                        Eigen::Vector3f norm2 = normals[index2];
                        float angle = norm1[0]*norm2[0]+norm1[1]*norm2[1]+norm1[2]*norm2[2];
                        if(angle<min) min=angle;
                    }
                }

                //check if within threshold for feature detection
                if(min>=threshold) continue;

                //set feature points to 1.0
                for(int i_=0; i_<2; i_++){
                    for(int j_=0; j_<2; j_++){
                        for(int k_=0; k_<2; k_++){
                            if((*surfaceMap)[i+i_][j+j_][k+k_]>-1.0){
                                (*featureMap)[i+i_][j+j_][k+k_]=1.0;
                            }
                        }
                    }
                }

            }
        }
    }//end of iteration through grid

    return featureMap;
}

