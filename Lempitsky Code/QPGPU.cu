#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <memory.h>
#include "mex.h"
                     

texture<float> inTex;

__global__ void ProcessRow(float *out,
    int *ir, float *pr, float *invdg,
    float *lb, float *ub, int nRows, int nCols)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;

    if(row < nRows) {
        float res = 0;
        int i;

        for(i = 0; i < nCols; i++) 
        {
            int a = nRows*i+row;
            float val = pr[a];
            float x = tex1Dfetch(inTex, ir[a]);
            if(val)
                res +=  val*x;
        }

        res = (tex1Dfetch(inTex, row)-res*invdg[row])*0.5f;
        if(res < lb[row]) res = lb[row];
        if(res > ub[row]) res = ub[row];
        out[row] = res;
    }
}


//nvmex -f nvmexopts.bat QPGPU.cu -Ic:\cuda\include -Lc:\cuda\lib -lcudart

void mexFunction(
		 int		nlhs,
		 mxArray	*plhs[],
		 int		nrhs,
		 const mxArray	*prhs[]
		 )
{
    if (nrhs != 7) 
      mexErrMsgIdAndTxt("MATLAB:WrongNumberOfInputs", "7 input arguments required");
    if (nlhs != 1) 
      mexErrMsgIdAndTxt("MATLAB:WrongNumberOfOutputs", "1 output argument required");
    int *ir = mxGetIr(prhs[0]);
    int *jc = mxGetJc(prhs[0]);
    double *pr = mxGetPr(prhs[0]);
    int size = mxGetM(prhs[0]);
    
    float *invdg = (float *)mxGetData(prhs[1]);
    float *lb = (float *)mxGetData(prhs[2]);
    float *ub = (float *)mxGetData(prhs[3]);
    float *x0 = (float *)mxGetData(prhs[4]);
    int iter = int(*mxGetPr(prhs[5]));

    float *prSingle;
    int *ir_;
    int ncols;

    ncols = (int)*mxGetPr(prhs[6]);
    prSingle = new float[ncols*size];
    ir_ = new int[ncols*size];
    memset(prSingle, 0, sizeof(float)*ncols*size);
    memset(ir_, 0, sizeof(int)*ncols*size);
    for(int i = 0; i < size; i++)
        for(int j = jc[i], k = 0; j < jc[i+1]; j++, k++)
        {
            prSingle[k*size+i] = (float)pr[j]; 
            ir_[k*size+i] = ir[j];
        }
    

    int *irCu;
    float *prCu, *invdgCu, *lbCu, *ubCu, *inCu, *outCu;

    int sizeMat = size*ncols;

    hipMalloc((void**)&irCu, sizeMat * sizeof(int));
    hipMalloc((void**)&prCu, sizeMat * sizeof(float));  
    hipMalloc((void**)&invdgCu, size * sizeof(float));  
    hipMalloc((void**)&lbCu, size * sizeof(float));  
    hipMalloc((void**)&ubCu, size * sizeof(float));  
    hipMalloc((void**)&inCu, size * sizeof(float));  
    hipMalloc((void**)&outCu, size * sizeof(float));  

    hipMemcpy(irCu, ir_, sizeMat * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(prCu, prSingle, sizeMat * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(invdgCu, invdg, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(lbCu, lb, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ubCu, ub, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inCu, x0, size * sizeof(float), hipMemcpyHostToDevice);

   
    int BLOCK_SIZE = 256;
    int NBLOCKS = (int)ceil(double(size)/BLOCK_SIZE);

    hipBindTexture(0, inTex, inCu, size*sizeof(float));

    clock_t startTime = clock();

    for(int i = 0; i < iter; i++)
    {
        ProcessRow<<<NBLOCKS,BLOCK_SIZE>>>(outCu, irCu, prCu, invdgCu, lbCu, ubCu, size, ncols);                                            
        hipMemcpy(inCu, outCu, size * sizeof(float), hipMemcpyDeviceToDevice);
    }
    hipDeviceSynchronize();
    mexPrintf("Elapsed time for the iterations = %lf\n", (double(clock())-startTime)/CLOCKS_PER_SEC);    
        
    int dims[] = {size,1};
    plhs[0] = mxCreateNumericArray( 2, dims, mxSINGLE_CLASS, mxREAL );
    float *result = (float *)mxGetData(plhs[0]);
    hipMemcpy(result, outCu, size * sizeof(float), hipMemcpyDeviceToHost);
    
    delete ir_;
    delete prSingle;
    hipFree(irCu);
    hipFree(prCu);
    hipFree(invdgCu);
    hipFree(lbCu);
    hipFree(ubCu);
    hipFree(inCu);
    hipFree(outCu);
}